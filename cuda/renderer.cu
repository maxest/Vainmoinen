
#include <hip/hip_runtime.h>
#ifdef VRENDERER_CUDA

#include <vainmoinen/renderer.hpp>



cudaStream_t stream1, stream2;

cudaGraphicsResource* colorBuffer_cudaResource;
uchar4 *colorBuffer;

float *depthBuffer;

#ifndef USE_CONSTANT_MEMORY
	TriangleToRasterize_CUDA *trianglesToRasterizeBuffer;
#else
	__constant__ TriangleToRasterize_CUDA trianglesToRasterizeBuffer[256];
#endif
uint *indicesToTrianglesToRasterizeBuffer_CPU;
uint *indicesToTrianglesToRasterizeBuffer;



void CVRenderer::create_CUDA()
{
	cudaDeviceProp prop;
	int device;

	memset(&prop, 0, sizeof(cudaDeviceProp));
	prop.major = 1;
	prop.minor = 0;
	cudaChooseDevice(&device, &prop);

	cudaGLSetGLDevice(device);

	glewInit();

	//

	cudaStreamCreate(&stream1);
	cudaStreamCreate(&stream2);

	glGenBuffers(1, &colorBuffer_GL_pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, colorBuffer_GL_pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, 4 * screenWidth * screenHeight, NULL, GL_STREAM_DRAW);

	cudaGraphicsGLRegisterBuffer(&colorBuffer_cudaResource, colorBuffer_GL_pbo, cudaGraphicsMapFlagsNone);

	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &colorBuffer_GL_texture);
	glBindTexture(GL_TEXTURE_2D, colorBuffer_GL_texture);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, screenWidth, screenHeight, 0, GL_BGRA, GL_UNSIGNED_BYTE, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	cudaMalloc((void**)&depthBuffer, 4 * screenWidth * screenHeight);

	#ifndef USE_CONSTANT_MEMORY
		cudaMalloc((void**)&trianglesToRasterizeBuffer, TRIANGLES_PER_PACK * sizeof(TriangleToRasterize));
	#endif
	cudaHostAlloc((void**)&indicesToTrianglesToRasterizeBuffer_CPU, screenWidth_tiles * screenHeight_tiles * TRIANGLES_PER_PACK * sizeof(uint), cudaHostAllocDefault);
	cudaMalloc((void**)&indicesToTrianglesToRasterizeBuffer, screenWidth_tiles * screenHeight_tiles * TRIANGLES_PER_PACK * sizeof(uint));
}



void CVRenderer::destroy_CUDA()
{
	#ifndef USE_CONSTANT_MEMORY
		cudaFree(trianglesToRasterizeBuffer);
	#endif
	cudaFreeHost(indicesToTrianglesToRasterizeBuffer_CPU);
	cudaFree(indicesToTrianglesToRasterizeBuffer);

	cudaFree(depthBuffer);

	glBindTexture(GL_TEXTURE_2D, colorBuffer_GL_texture);
	glDeleteTextures(1, &colorBuffer_GL_texture);

	cudaGraphicsUnregisterResource(colorBuffer_cudaResource);

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, colorBuffer_GL_pbo);
	glDeleteBuffers(1, &colorBuffer_GL_pbo);

	cudaStreamDestroy(stream1);
	cudaStreamDestroy(stream2);
}



__global__ void clearBuffers(int screenWidth, uchar4 *colorBuffer, float *depthBuffer)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int index = x + y*screenWidth;

	colorBuffer[index].x = 0;
	colorBuffer[index].y = 0;
	colorBuffer[index].z = 0;
	colorBuffer[index].w = 255;

	depthBuffer[index] = 1.0f;
}



#ifndef USE_CONSTANT_MEMORY
	__global__ void rasterize(int screenWidth, int screenWidth_tiles, int y_offset, uchar4 *colorBuffer, float *depthBuffer, TriangleToRasterize_CUDA *trianglesToRasterizeBuffer, uint *indicesToTrianglesToRasterizeBuffer)
#else
	__global__ void rasterize(int screenWidth, int screenWidth_tiles, int y_offset, uchar4 *colorBuffer, float *depthBuffer, uint *indicesToTrianglesToRasterizeBuffer)
#endif
{
	int pixelX = threadIdx.x + blockIdx.x*blockDim.x;
	int pixelY = threadIdx.y + blockIdx.y*blockDim.y + y_offset;
	int pixelIndex = pixelX + pixelY*screenWidth;
	int tileIndex = blockIdx.x + screenWidth_tiles*(blockIdx.y + y_offset/16);

	if (pixelX >= screenWidth)
		return;

	for (int i = 0; i < TRIANGLES_PER_PACK; i++)
	{
		int triangleIndex = indicesToTrianglesToRasterizeBuffer[TRIANGLES_PER_PACK*tileIndex + i];

		if (triangleIndex == 0)
			break;

		TriangleToRasterize_CUDA& t = trianglesToRasterizeBuffer[triangleIndex - 1];

		// we want centers of pixels to be used in computations
		float x = (float)pixelX + 0.5f;
		float y = (float)pixelY + 0.5f;

		// affine barycentric weights
		float alpha = implicitLine_device(x, y, t.v1.position, t.v2.position) * t.one_over_v0ToLine12;
		float beta = implicitLine_device(x, y, t.v2.position, t.v0.position) * t.one_over_v1ToLine20;
		float gamma = implicitLine_device(x, y, t.v0.position, t.v1.position) * t.one_over_v2ToLine01;

		// if pixel (x, y) is inside the triangle or on one of its edges
		if (alpha >= 0 && beta >= 0 && gamma >= 0)
		{
			float z_affine = alpha*t.v0.position.z + beta*t.v1.position.z + gamma*t.v2.position.z;

			if (z_affine < depthBuffer[pixelIndex] && z_affine <= 1.0f)
			{
				// perspective-correct barycentric weights
				float l = alpha*t.one_over_z0 + beta*t.one_over_z1 + gamma*t.one_over_z2;
				l = 1.0f / l;
				alpha *= l*t.one_over_z0;
				beta *= l*t.one_over_z1;
				gamma *= l*t.one_over_z2;

				// attributes interpolation
				vec3_CUDA color_persp = alpha*t.v0.color + beta*t.v1.color + gamma*t.v2.color;

				// run pixel shader
				vec3_CUDA pixelColor = color_persp;

				// clamp bytes to 255
				byte red = (byte)(255.0f * MIN(pixelColor.x, 1.0f));
				byte green = (byte)(255.0f * MIN(pixelColor.y, 1.0f));
				byte blue = (byte)(255.0f * MIN(pixelColor.z, 1.0f));

				// update buffers
				colorBuffer[pixelIndex].x = red;
				colorBuffer[pixelIndex].y = green;
				colorBuffer[pixelIndex].z = blue;
				depthBuffer[pixelIndex] = z_affine;
			}
		}
	}
}



void CVRenderer::runPixelProcessor_CUDA()
{
	size_t size;
	cudaGraphicsMapResources(1, &colorBuffer_cudaResource, NULL);
	cudaGraphicsResourceGetMappedPointer((void**)&colorBuffer, &size, colorBuffer_cudaResource);

	// clear buffers
	dim3 blocks(screenWidth_tiles, screenHeight_tiles);
	dim3 threads(16, 16);
	clearBuffers<<<blocks, threads>>>(screenWidth, colorBuffer, depthBuffer);

	if (trianglesToRasterize.size() == 0)
	{
		cudaGraphicsUnmapResources(1, &colorBuffer_cudaResource, NULL);
		return;
	}

	int packsNum = (trianglesToRasterize.size() - 1) / TRIANGLES_PER_PACK + 1;
	int remainingTrianglesToRasterizeNum = trianglesToRasterize.size();

	for (int k = 0; k < packsNum; k++)
	{
		int trianglesToRasterizeNumInCurrentPack;

		if (remainingTrianglesToRasterizeNum >= TRIANGLES_PER_PACK)
		{
			trianglesToRasterizeNumInCurrentPack = TRIANGLES_PER_PACK;
			remainingTrianglesToRasterizeNum -= TRIANGLES_PER_PACK;
		}
		else
		{
			trianglesToRasterizeNumInCurrentPack = remainingTrianglesToRasterizeNum;
		}

		memset(indicesToTrianglesToRasterizeBuffer_CPU, 0, screenWidth_tiles * screenHeight_tiles * TRIANGLES_PER_PACK * sizeof(uint));

		for (int i = 0; i < screenWidth_tiles * screenHeight_tiles; i++)
		{
			int tileX = i % screenWidth_tiles;
			int tileY = i / screenWidth_tiles;
			int offset = 0;

			for (int j = 0; j < trianglesToRasterizeNumInCurrentPack; j++)
			{
				if (trianglesToRasterize[TRIANGLES_PER_PACK*k + j].coversTile(tileX, tileY))
				{
					indicesToTrianglesToRasterizeBuffer_CPU[TRIANGLES_PER_PACK*i + offset] = j + 1;
					offset++;
				}
			}
		}

		//

		// copy trianglesToRasterizeBuffer to GPU
		#ifndef USE_CONSTANT_MEMORY
			cudaMemcpy(trianglesToRasterizeBuffer, &trianglesToRasterize[TRIANGLES_PER_PACK*k], trianglesToRasterizeNumInCurrentPack * sizeof(TriangleToRasterize), cudaMemcpyHostToDevice);
		#else
			cudaMemcpyToSymbol(trianglesToRasterizeBuffer, &trianglesToRasterize[TRIANGLES_PER_PACK*k], trianglesToRasterizeNumInCurrentPack * sizeof(TriangleToRasterize));
		#endif

		#ifndef STREAMS_FREQUENCY_DIVIDER
			// copy indicesToTrianglesToRasterizeBuffer to GPU
			cudaMemcpy(indicesToTrianglesToRasterizeBuffer, indicesToTrianglesToRasterizeBuffer_CPU, screenWidth_tiles * screenHeight_tiles * TRIANGLES_PER_PACK * sizeof(uint), cudaMemcpyHostToDevice);

			// clear buffers and rasterize
			dim3 blocks(screenWidth_tiles, screenHeight_tiles);
			dim3 threads(16, 16);
			#ifndef USE_CONSTANT_MEMORY
				rasterize<<<blocks, threads>>>(screenWidth, screenWidth_tiles, 0, colorBuffer, depthBuffer, trianglesToRasterizeBuffer, indicesToTrianglesToRasterizeBuffer);
			#else
				rasterize<<<blocks, threads>>>(screenWidth, screenWidth_tiles, 0, colorBuffer, depthBuffer, indicesToTrianglesToRasterizeBuffer);
			#endif
		#else
			for (int i = 0; i < screenHeight_tiles/STREAMS_FREQUENCY_DIVIDER; i += 2)
			{
				int offset1 = i * screenWidth_tiles * TRIANGLES_PER_PACK * STREAMS_FREQUENCY_DIVIDER;
				int offset2 = (i+1) * screenWidth_tiles * TRIANGLES_PER_PACK * STREAMS_FREQUENCY_DIVIDER;

				// copy indicesToTrianglesToRasterizeBuffer to GPU
				cudaMemcpyAsync(indicesToTrianglesToRasterizeBuffer + offset1, indicesToTrianglesToRasterizeBuffer_CPU + offset1, STREAMS_FREQUENCY_DIVIDER * screenWidth_tiles * TRIANGLES_PER_PACK * sizeof(uint), cudaMemcpyHostToDevice, stream1);
				cudaMemcpyAsync(indicesToTrianglesToRasterizeBuffer + offset2, indicesToTrianglesToRasterizeBuffer_CPU + offset2, STREAMS_FREQUENCY_DIVIDER * screenWidth_tiles * TRIANGLES_PER_PACK * sizeof(uint), cudaMemcpyHostToDevice, stream2);

				// rasterize
				dim3 blocks(screenWidth_tiles, STREAMS_FREQUENCY_DIVIDER);
				dim3 threads(16, 16);
				#ifndef USE_CONSTANT_MEMORY
					rasterize<<<blocks, threads, 0, stream1>>>(screenWidth, screenWidth_tiles, STREAMS_FREQUENCY_DIVIDER * 16 * i, colorBuffer, depthBuffer, trianglesToRasterizeBuffer, indicesToTrianglesToRasterizeBuffer);
					rasterize<<<blocks, threads, 0, stream2>>>(screenWidth, screenWidth_tiles, STREAMS_FREQUENCY_DIVIDER * 16 * (i+1), colorBuffer, depthBuffer, trianglesToRasterizeBuffer, indicesToTrianglesToRasterizeBuffer);
				#else
					rasterize<<<blocks, threads, 0, stream1>>>(screenWidth, screenWidth_tiles, STREAMS_FREQUENCY_DIVIDER * 16 * i, colorBuffer, depthBuffer, indicesToTrianglesToRasterizeBuffer);
					rasterize<<<blocks, threads, 0, stream2>>>(screenWidth, screenWidth_tiles, STREAMS_FREQUENCY_DIVIDER * 16 * (i+1), colorBuffer, depthBuffer, indicesToTrianglesToRasterizeBuffer);
				#endif
			}

			cudaStreamSynchronize(stream1);
			cudaStreamSynchronize(stream2);
		#endif
	}

	cudaGraphicsUnmapResources(1, &colorBuffer_cudaResource, NULL);
}



#endif
